#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template<typename Dtype>
__global__ void MaxUnPoolForward(const int nthreads, const Dtype* bottom_data,
        const int num, const int channels, const int height, const int width,
        const int unpooled_height, const int unpooled_width, const int kernel_h,
        const int kernel_w, const int stride_h, const int stride_w,
        const int pad_h, const int pad_w, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads)
    {
        int pw = index % width;
        int ph = (index / width) % height;
        int c = (index / width / height) % channels;
        int n = index / width / height / channels;

        int uph = ph * stride_h;
        int upw = pw * stride_w;
        int unpooled_index = uph * unpooled_width + upw;

        top_data += (n * channels + c) * unpooled_height * unpooled_width;
        top_data[unpooled_index] = bottom_data[index];
    }
}

template<typename Dtype>
__global__ void AveUnPoolForward(const int nthreads, const Dtype* bottom_data,
        const int num, const int channels, const int unpooled_height,
        const int unpooled_width, const int height, const int width,
        const int kernel_h, const int kernel_w, const int stride_h,
        const int stride_w, const int pad_h, const int pad_w, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads)
    {
        // find out the local index
        // find out the local offset
        int w = index % unpooled_width + pad_w;
        int h = (index / unpooled_width) % unpooled_height + pad_h;
        int c = (index / unpooled_width / unpooled_height) % channels;
        int n = index / unpooled_width / unpooled_height / channels;

        int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
        int phend = min(h / stride_h + 1, height);
        int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
        int pwend = min(w / stride_w + 1, width);
        Dtype distval = 0;
        bottom_data += (n * channels + c) * height * width;
        for (int ph = phstart; ph < phend; ++ph) {
            for (int pw = pwstart; pw < pwend; ++pw) {
                // figure out the pooling size
                int hstart = ph * stride_h - pad_h;
                int wstart = pw * stride_w - pad_w;
                int hend = min(hstart + kernel_h, unpooled_height + pad_h);
                int wend = min(wstart + kernel_w, unpooled_width + pad_w);
                int pool_size = (hend - hstart) * (wend - wstart);
                distval += bottom_data[ph * width + pw] / pool_size;
            }
        }
        top_data[index] = distval;
    }
}

template<typename Dtype>
void UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
        const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    int count = bottom[0]->count();
    Dtype* top_data = top[0]->mutable_gpu_data();
    caffe_gpu_set(top[0]->count(), Dtype(0.), top_data);

    // here we don't use mask
    // const bool use_bottom_mask = false;

    switch (this->layer_param_.unpooling_param().unpool()) {
    case UnPoolingParameter_UnPoolMethod_MAX:

        // NOLINT_NEXT_LINE(whitespace/operators)
        MaxUnPoolForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, bottom[0]->num(),
                channels_, height_, width_, unpooled_height_, unpooled_width_,
                kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
                top_data);
        break;

    case UnPoolingParameter_UnPoolMethod_AVE:
        // NOLINT_NEXT_LINE(whitespace/operators)
        AveUnPoolForward<Dtype> <<<CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS>>>(top[0]->count(), bottom_data,
                bottom[0]->num(), channels_, unpooled_height_, unpooled_width_,
                height_, width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
                pad_h_, pad_w_, top_data);
        break;
    default:
        LOG(FATAL)<< "Unknown unpooling method.";
    }
    CUDA_POST_KERNEL_CHECK
    ;
}

template<typename Dtype>
__global__ void MaxUnPoolBackward(const int nthreads, const Dtype* top_diff,
        const int num, const int channels, const int height, const int width,
        const int unpooled_height, const int unpooled_width, const int kernel_h,
        const int kernel_w, const int stride_h, const int stride_w,
        const int pad_h, const int pad_w, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads)
    {
        // find out the local index
        // find out the local offset
        int pw = index % width;
        int ph = (index / width) % height;
        int c = (index / width / height) % channels;
        int n = index / width / height / channels;

        // here be more strict
        // because pad == 0
        // && height_ * stride + kernel <= unpooled_height_
        // we put the point at left up corner
        int uph = ph * stride_h;
        int upw = pw * stride_w;
        int unpooled_index = uph * unpooled_width + upw;

        top_diff += (n * channels + c) * unpooled_height * unpooled_width;
        bottom_diff[index] = top_diff[unpooled_index];
    }
}

template<typename Dtype>
__global__ void AveUnPoolBackward(const int nthreads, const Dtype* top_diff,
        const int num, const int channels, const int unpooled_height,
        const int unpooled_width, const int height, const int width,
        const int kernel_h, const int kernel_w, const int stride_h,
        const int stride_w, const int pad_h, const int pad_w,
        Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads)
    {
        int pw = index % width;
        int ph = (index / width) % height;
        int c = (index / width / height) % channels;
        int n = index / width / height / channels;
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, unpooled_height + pad_h);
        int wend = min(wstart + kernel_w, unpooled_width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        hend = min(hend, unpooled_height);
        wend = min(wend, unpooled_width);
        Dtype gradient = 0;
        top_diff += (n * channels + c) * unpooled_height * unpooled_width;
        for (int h = hstart; h < hend; ++h) {
            for (int w = wstart; w < wend; ++w) {
                gradient += top_diff[h * unpooled_width + w];
            }
        }
        bottom_diff[index] = gradient / pool_size;
    }
}

template<typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down,
        const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
        return;
    }
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    caffe_gpu_set(count, Dtype(0.), bottom_diff);

    // disable mask
    // const bool use_bottom_mask = false;

    switch (this->layer_param_.unpooling_param().unpool()) {
    case UnPoolingParameter_UnPoolMethod_MAX:

        // NOLINT_NEXT_LINE(whitespace/operators)
        MaxUnPoolBackward<Dtype> <<<CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, top[0]->num(),
                channels_, height_, width_, unpooled_height_, unpooled_width_,
                kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
                bottom_diff);
        break;

    case UnPoolingParameter_UnPoolMethod_AVE:
        // NOLINT_NEXT_LINE(whitespace/operators)
        AveUnPoolBackward<Dtype> <<<CAFFE_GET_BLOCKS(count),
                CAFFE_CUDA_NUM_THREADS>>>(bottom[0]->count(), top_diff,
                top[0]->num(), channels_, unpooled_height_, unpooled_width_,
                height_, width_, kernel_h_, kernel_w_, stride_h_, stride_w_,
                pad_h_, pad_w_, bottom_diff);
        break;
    default:
        LOG(FATAL)<< "Unknown unpooling method.";
    }
    CUDA_POST_KERNEL_CHECK
    ;
}

INSTANTIATE_LAYER_GPU_FUNCS(UnPoolingLayer);

} // namespace caffe
