#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template<typename Dtype>
__global__ void MultinomialLogisticLossForwardGPU(const int nthreads,
        const Dtype* data, const Dtype* label, Dtype* loss, const int num,
        const int dim) {

    CUDA_KERNEL_LOOP(index, nthreads)
    {

        const int n = index / dim;
        const int s = index % dim;

        const Dtype label_value = label[n * dim + s];
        const Dtype prob = max(data[n * dim + s], Dtype(FLT_MIN));
        loss[n * dim + s] = -label_value * log(prob);
    }
}

template<typename Dtype>
void MultinomialLogisticLossLayer<Dtype>::Forward_gpu(
        const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_label = bottom[1]->gpu_data();
    const int num = bottom[0]->num();
    const int dim = bottom[0]->count() / bottom[0]->num();
    const int nthreads = num * dim;

    // Since this memory is not used for anything until it is overwritten
    // on the backward pass, we use it here to avoid having to allocate new GPU
    // memory to accumulate intermediate results in the kernel.
    Dtype* loss_data = bottom[0]->mutable_gpu_diff();
    caffe_gpu_set(bottom[0]->count(), Dtype(0), loss_data);

    // NOLINT_NEXT_LINE(whitespace/operators)
    MultinomialLogisticLossForwardGPU<Dtype> <<<CAFFE_GET_BLOCKS(nthreads),
            CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, bottom_label,
            loss_data, num, dim);

    Dtype loss;
    caffe_gpu_asum(nthreads, loss_data, &loss);
    loss = loss / nthreads;

    top[0]->mutable_cpu_data()[0] = loss;
}

template<typename Dtype>
__global__ void MultinomialLogisticLossBackwardGPU(const int nthreads,
        const Dtype* data, const Dtype* label, Dtype* bottom_diff,
        const int num, const int dim, const Dtype scale) {

    CUDA_KERNEL_LOOP(index, nthreads)
    {

        const int n = index / dim;
        const int s = index % dim;

        const Dtype label_value = label[n * dim + s];
        const Dtype prob = max(data[n * dim + s], Dtype(FLT_MIN));

        bottom_diff[n * dim + s] = scale * label_value / prob;
    }
}

template<typename Dtype>
void MultinomialLogisticLossLayer<Dtype>::Backward_gpu(
        const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
        const vector<Blob<Dtype>*>& bottom) {

    bool ispropagationdown = true;
    if (is_grl_train) {
        if (is_grl_loss == 2) {
            ispropagationdown = false;
            if (propagate_down[0]) {
                Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
                caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
            }
        }
    } else {
        if (is_grl_loss == 1) {
            ispropagationdown = false;
            if (propagate_down[0]) {
                Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
                caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
            }
        }
    }

    // Backward_cpu(top, propagate_down, bottom);
    if (propagate_down[1]) {
        LOG(FATAL)<< this->type()
        << " Layer cannot backpropagate to label inputs.";
    }

    if (propagate_down[0] && ispropagationdown) {
        const Dtype* bottom_data = bottom[0]->gpu_data();
        const Dtype* bottom_label = bottom[1]->gpu_data();

        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
        caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);

        const int num = bottom[0]->num();
        const int dim = bottom[0]->count() / bottom[0]->num();
        const int nthreads = num * dim;

        const Dtype scale = -top[0]->cpu_diff()[0] / nthreads;

        MultinomialLogisticLossBackwardGPU<Dtype> <<<CAFFE_GET_BLOCKS(nthreads),
                CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_data, bottom_label,
                bottom_diff, num, dim, scale);
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(MultinomialLogisticLossLayer);

} // namespace caffe
